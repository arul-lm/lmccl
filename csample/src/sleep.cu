
#include <hip/hip_runtime.h>
#include <time.h>

// CUDA kernel to pause for at least num_cycle cycles
__global__ void sleep(int64_t num_cycles)
{
    int64_t cycles = 0;
    int64_t start = clock64();
    while(cycles < num_cycles) {
        cycles = clock64() - start;
    }
}

// Launches a kernel that sleeps for num_cycles
extern "C" void sleep_kernel(int64_t num_cycles, hipStream_t stream)
{
    // Launch a single GPU thread to sleep
    int blockSize, gridSize;
    blockSize = 1;
    gridSize = 1;
 
    // Execute the kernel
    sleep<<< gridSize, blockSize, 0, stream >>>(num_cycles);
}